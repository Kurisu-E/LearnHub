#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>

#include "utils.cuh"

#define ROWS 2500
#define COLS 1000

#if ROWS * COLS <= 2000
#define DATA_PRINT
#endif

FILE* fp;

__global__ void _kernel_matrixVectorMul(int* matrix, int* vector, int* result,
                                        int rows, int cols) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < rows) {
    int sum = 0;
    for (int j = 0; j < cols; ++j)
      sum += matrix[idx * cols + j] * vector[j];
    result[idx] = sum;
  }
}

/// @brief 矩阵向量乘法并行实现
/// @return 执行时间（ms）
float matrixVectorMulParallel(int* h_matrix, int* h_vector, int* h_result,
                              int rows, int cols) {
  size_t matrix_size = rows * cols * sizeof(int);
  size_t vector_size = cols * sizeof(int);
  size_t result_size = rows * sizeof(int);

  int* d_matrix,
     * d_vector,
     * d_result;

  CUDA_CHECK(hipMalloc(&d_matrix, matrix_size));
  CUDA_CHECK(hipMalloc(&d_vector, vector_size));
  CUDA_CHECK(hipMalloc(&d_result, result_size));

  CUDA_CHECK(hipMemcpy(d_matrix, h_matrix, matrix_size, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_vector, h_vector, vector_size, hipMemcpyHostToDevice));

  dim3 block(512);
  dim3 grid((rows - 1) / block.x + 1);

  float time = KERNEL_TIMER(_kernel_matrixVectorMul, false, grid, block,
                            d_matrix, d_vector, d_result, rows, cols);

  CUDA_CHECK(hipMemcpy(h_result, d_result, result_size, hipMemcpyDeviceToHost));

  CUDA_CHECK(hipFree(d_matrix));
  CUDA_CHECK(hipFree(d_vector));
  CUDA_CHECK(hipFree(d_result));
  return time;
}

/// @brief 矩阵向量乘法串行实现
/// @return 执行时间（ms）
float matrixVectorMulSerial(int* matrix, int* vector, int* result,
                            int rows, int cols) {
  clock_t start, end;
  start = clock();
  for (int i = 0; i < rows; ++i) {
    int sum = 0;
    for (int j = 0; j < cols; ++j)
      sum += matrix[i * cols + j] * vector[j];
    result[i] = sum;
  }
  end = clock();
  return (float)(end - start) / CLOCKS_PER_SEC * 1000;
}

bool matchResult(int* a, int* b, int size) {
  for (int i = 0; i < size; ++i)
    if (a[i] != b[i])
      return false;
  return true;
}

int main() {
  srand(time(NULL));
  int* h_matrix = (int*)malloc(ROWS * COLS * sizeof(int));
  int* h_vector = (int*)malloc(COLS * sizeof(int));
  int* h_result = (int*)malloc(ROWS * sizeof(int));
  for (int i = 0; i < ROWS; ++i) {
    for (int j = 0; j < COLS; ++j) {
      h_matrix[i * COLS + j] = rand() % 11 - 5; // 随机化矩阵元素（-5 ~ 5）
      h_vector[j] = rand() % 11 - 5;            // 随机化向量元素（-5 ~ 5）
    }
  }

  fp = fopen("output.txt", "w");
  if (fp == NULL)
    perror("Failed to open file");
#ifdef DATA_PRINT
  dual_printf("Matrix:\n");
  print_matrix(h_matrix, ROWS, COLS);
  dual_printf("Vector:\n");
  print_vector(h_vector, COLS);
#endif

  float time_parallel = matrixVectorMulParallel(h_matrix, h_vector, h_result,
                                                ROWS, COLS);
#ifdef DATA_PRINT
  dual_printf("Parallel result:\n");
  print_vector(h_result, ROWS);
#endif

  int* result = (int*)malloc(ROWS * sizeof(int));
  float time_serial = matrixVectorMulSerial(h_matrix, h_vector, result,
                                            ROWS, COLS);
#ifdef DATA_PRINT
  dual_printf("Serial result:\n");
  print_vector(result, ROWS);
#endif

  if (matchResult(h_result, result, ROWS)) dual_printf("Result match!\n");
  else dual_printf("Result mismatch!\n");

  dual_printf("Parallel time: %f ms\n", time_parallel);
  dual_printf("Serial time: %f ms\n", time_serial);

  fclose(fp);
  free(h_matrix);
  free(h_vector);
  free(h_result);
  free(result);
  return 0;
}
